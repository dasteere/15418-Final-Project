#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdio.h>
#include <limits.h>
#include <assert.h>
#include <time.h>

extern "C" {
#include "cudaRiver.h"
}

/*struct GlobalConstants {
    //must be in cuda
    int *oopRanks;
    int oopSize;

    //must be in cuda
    int *ipRanks;
    int ipSize;
    int potSize;
    int betSize;
    int afterBetSize;
};*/

__constant__ GlobalConstants cuConsts;

/*#define CHECK_CALL 0
#define CHECK_FOLD 1
#define BET 2
#define OOP_MOVES 3

#define IP_MOVES 4

#define NUM_STRATEGIES_PER_ITERATION 100*/

char **cudaOopStrategies;
int *cudaOutput;
int *output;
int *cudaPows;

__device__ void setStrategy(char *strategy, int start, int *pows) {
    int i = cuConsts.oopSize - 1;
    int toAdd = 0;
    while (start > 0 && i > 0) {
        toAdd = start / pows[i];
        start -= toAdd;
        strategy[i] = toAdd;
        i--;
    }
    if (i < 0) assert(0);
}

__device__ void addOne(char *curStrategy) {
    for (int i = 0; i < cuConsts.oopSize; i++) {
        curStrategy[i] = (curStrategy[i] + 1) % OOP_MOVES;
        if (curStrategy[i] != 0) break;
    }
}

//TODO: maybe use extern shared for the curStrategy and the max value
__global__ void kernel_findBestOopStrat(int numThreads, int numBlocks,
        int numStrategiesPerBlock, int totalStrategies,
        char **outputStrategy, int *outputValue, int *pows) {
    int idx = threadIdx.x;
    int block = blockIdx.x;
    int startStrategy = numStrategiesPerBlock * numBlocks;
    // we will never have an oopSize of more than 100
    char strategy[16];

    setStrategy(strategy, startStrategy, pows);

    int maxStrategy = max(startStrategy + numStrategiesPerBlock, totalStrategies);

    int check = 0;
    int bet = 0;
    int call = 0;
    int fold = 0;
    int curMax = 0;
    int ipRank, oopRank, oopMove, showdown, showPot, showBet;
    for (int m = startStrategy; m < maxStrategy; m++) {
        if (startStrategy + m > totalStrategies) break;
        for (int i = idx; i < cuConsts.ipSize; i += numThreads) {
            ipRank = cuConsts.ipRanks[i];
            for (int j = 0; j < cuConsts.oopSize; j++) {
                oopRank = cuConsts.oopRanks[j];
                oopMove = strategy[j];
                showdown = ipRank > oopRank ? 1 : -1;
                showPot = ipRank > oopRank ? cuConsts.potSize : 0;
                showBet = showPot + (showdown * cuConsts.betSize);
                switch (oopMove) {
                case CHECK_CALL:
                    check += showPot;
                    bet += showBet;
                case CHECK_FOLD:
                    check += showPot;
                    bet += cuConsts.potSize;
                case BET:
                    call += showBet;
                }
            }
            atomicAdd(outputValue + block, max(check,bet) + max(call, fold));

            check = 0;
            bet = 0;
            call = 0;
        }

        __syncthreads();
        if (outputValue[block] > curMax) {
            curMax = outputValue[block];
            for (int i = idx; i < cuConsts.oopSize; i+= numThreads) {
                if (strategy[i] >= OOP_MOVES || strategy[i] < 0) assert(0);
                outputStrategy[block][i] = strategy[i];
            }
        }
        addOne(strategy);
    }

}

__global__ void kernel_calculateValue(int numThreads, int numBlocks,
        char **cudaOopStrategies, int *output) {
    int idx = threadIdx.x;
    int strategyIdx = blockIdx.x;

    int check = 0;
    int bet = 0;
    int call = 0;
    int fold = 0;
    int cb_max = 0;
    int cf_max = 0;
    int ipRank, oopRank, oopMove, showdown, showPot, showBet;
    for (int k = strategyIdx; k < NUM_STRATEGIES_PER_ITERATION; k += numBlocks) {
        char *strategy = cudaOopStrategies[k];
        for (int i = idx; i < cuConsts.ipSize; i += numThreads) {
            ipRank = cuConsts.ipRanks[i];
            for (int j = 0; j < cuConsts.oopSize; j++) {
                oopRank = cuConsts.oopRanks[j];
                oopMove = strategy[j];
                showdown = ipRank > oopRank ? 1 : -1;
                showPot = ipRank > oopRank ? cuConsts.potSize : 0;
                showBet = showPot + (showdown * cuConsts.betSize);
                switch (oopMove) {
                case CHECK_CALL:
                    check += showPot;
                    bet += showBet;
                case CHECK_FOLD:
                    check += showPot;
                    bet += cuConsts.potSize;
                case BET:
                    call += showBet;
                }
            }
            cb_max = check > bet ? check : bet;
            cf_max = call > fold ? call : fold;
            check = cb_max + cf_max;
            //could be a bottleneck
            atomicAdd(output + k, cb_max + cf_max);
            check = 0;
            bet = 0;
            call = 0;
            fold = 0;
        }
    }
}

__global__ void kernel_calculateIpStrat(int numThreads,
        char *strategy, char *betStrategy, char *checkStrategy) {
    int idx = threadIdx.x;

    int check = 0;
    int bet = 0;
    int call = 0;
    int fold = 0;
    int ipRank, oopRank, oopMove, showdown, showPot, showBet;
    for (int i = idx; i < cuConsts.ipSize; i += numThreads) {
        ipRank = cuConsts.ipRanks[i];
        for (int j = 0; j < cuConsts.oopSize; j++) {
            oopRank = cuConsts.oopRanks[j];
            oopMove = strategy[j];
            showdown = ipRank > oopRank ? 1 : -1;
            showPot = ipRank > oopRank ? cuConsts.potSize : 0;
            showBet = showPot + (showdown * cuConsts.betSize);
            switch (oopMove) {
            case CHECK_CALL:
                check += showPot;
                bet += showBet;
            case CHECK_FOLD:
                check += showPot;
                bet += cuConsts.potSize;
            case BET:
                call += showBet;
            }
        }
        checkStrategy[i] = check > bet ? IP_CHECK : IP_BET;
        betStrategy[i] = call > fold ? IP_CALL : IP_FOLD;
        check = 0;
        bet = 0;
        call = 0;
        fold = 0;
    }
}

extern "C"
GlobalConstants *calcGlobalConsts(board_t board, hand_t *oopRange,
        int oopSize, hand_t *ipRange, int ipSize, int potSize, int betSize) {
    GlobalConstants *params = (GlobalConstants *) malloc(sizeof(GlobalConstants));
    int *oopRanks = (int *) malloc(oopSize * sizeof(int));
    int *ipRanks = (int *) malloc(ipSize * sizeof(int));
    for (int i = 0; i < oopSize; i++) {
        oopRanks[i] = rank_of(&board, &oopRange[i]);
    }
    for (int i = 0; i < ipSize; i++) {
        ipRanks[i] = rank_of(&board, &ipRange[i]);
    }
    if (hipMalloc(&(params->oopRanks), sizeof(int) * oopSize) != hipSuccess) {
        printf("Cuda malloc failed line 106\n");
        assert(0);
    }
    hipMalloc(&(params->ipRanks), sizeof(int) * ipSize);

    if (hipMemcpy(params->oopRanks, oopRanks, sizeof(int) * oopSize,
                hipMemcpyHostToDevice) != hipSuccess) {
        printf("Cuda Memcpy failed line 109\n");
        assert(0);
    }
    if (hipMemcpy(params->ipRanks, ipRanks, sizeof(int) * ipSize,
                hipMemcpyHostToDevice) != hipSuccess) {
        printf("Cuda Memcpy Failed line 113\n");
        assert(0);
    }

    params->oopSize = oopSize;
    params->ipSize = ipSize;
    params->potSize = potSize;
    params->betSize = betSize;
    params->afterBetSize = potSize + betSize;

    return params;
}

void addOne(char *curStrategy, GlobalConstants *params) {
    for (int i = 0; i < params->oopSize - 1; i++) {
        curStrategy[i] = (curStrategy[i] + 1) % OOP_MOVES;
        if (curStrategy[i] != 0) break;
    }
}

extern "C"
void calcMaxOopStrategy(char *bestStrat, int *stratVal, GlobalConstants *params) {
    if (hipMemcpyToSymbol(HIP_SYMBOL(cuConsts), params,
                sizeof(GlobalConstants)) != hipSuccess) {
        printf("cuda memcpy params failed\n");
        assert(0);
    }

    int pows[19];
    for (int i = 0; i < 19; i++) {
        if (i == 0) pows[i] = 1;
        else pows[i] = pows[i-1] * params->oopSize;
    }
    if (hipMalloc(&cudaPows, 19 * sizeof(int)) != hipSuccess) {
        printf("cuda malloc failed cudaPows\n");
        assert(0);
    }
    if (hipMemcpy(cudaPows, pows, 19 * sizeof(int),
                hipMemcpyHostToDevice) != hipSuccess) {
        printf("cuda memcpy failed cudaPows\n");
        assert(0);
    }

    int totalStrategies = 1;
    for (int i = 0; i < params->oopSize; i++) {
        totalStrategies *= OOP_MOVES;
    }

    int numThreads = MAX_THREADS > params->ipSize ? params->ipSize : MAX_THREADS;
    int numBlocks = MAX_BLOCKS < NUM_STRATEGIES_PER_ITERATION
        ? MAX_BLOCKS : NUM_STRATEGIES_PER_ITERATION;
    int strategiesPerBlock = totalStrategies / numBlocks;

    char **oopStrategies =
        (char **) malloc(numBlocks * sizeof(char *));
    if (hipMalloc(&cudaOopStrategies, numBlocks * sizeof(char *)) != hipSuccess) {
        printf("Cuda malloc failed cudaOopStrategies\n");
        assert(0);
    }

    for (int i = 0; i < numBlocks; i++) {
        if (hipMalloc(&oopStrategies[i],
                    params->oopSize * sizeof(char)) != hipSuccess) {
            printf("cuda malloc failed oopStrategies\n");
            assert(0);
        }
    }
    if (hipMemcpy(cudaOopStrategies, oopStrategies, numBlocks * sizeof(char*),
            hipMemcpyHostToDevice) != hipSuccess) {
        printf("cuda memcpy failed cudaOopStrategies\n");
        assert(0);
    }

    int *outputValues = (int *) malloc(numBlocks * sizeof(int));
    int *cudaOutputValues;
    if (hipMalloc(&cudaOutputValues, numBlocks * sizeof(int)) != hipSuccess) {
        printf("cuda malloc failed outputValues\n");
        assert(0);
    }

    kernel_findBestOopStrat<<<numBlocks, numThreads>>>(numThreads, numBlocks,strategiesPerBlock, totalStrategies, cudaOopStrategies, cudaOutputValues, cudaPows);

    hipDeviceSynchronize();

    if (hipMemcpy(outputValues, cudaOutputValues, numBlocks * sizeof(int), hipMemcpyDeviceToHost) != hipSuccess) {
        printf("Cuda memcpy failed outputValues\n");
        assert(0);
    }
    int maxIdx = 0;
    int max = -1;
    for (int i = 0; i < numBlocks; i++) {
        if (outputValues[i] > max) {
            max = outputValues[i];
            maxIdx = i;
        }
    }
    if (hipMemcpy(bestStrat, oopStrategies[maxIdx], params->oopSize * sizeof(char),
                hipMemcpyDeviceToHost) != hipSuccess) {
        printf("Cuda memcpy failed bestStrat\n");
        assert(0);
    }
    for (int i = 0; i < params->oopSize; i++) {
        printf("%d", bestStrat[i]);
    }
    printf("\n");
}
/*
//calculates the best strategy for the oop player along with the strategies value
extern "C"
void calcMaxStrategy(char *bestStrat, int *stratVal, GlobalConstants *params) {
    if (hipMemcpyToSymbol(HIP_SYMBOL(cuConsts), params,
                sizeof(GlobalConstants)) != hipSuccess) {
        printf("cuda memcpy to symbol failed line 141\n");
        assert(0);
    }


    char **oopStrategies =
        (char **) malloc(NUM_STRATEGIES_PER_ITERATION * sizeof(char *));
    if (hipMalloc(&cudaOopStrategies,
                NUM_STRATEGIES_PER_ITERATION * sizeof(char *)) != hipSuccess) {
        printf("Cuda malloc failed line 149\n");
        assert(0);
    }

    for (int i = 0; i < NUM_STRATEGIES_PER_ITERATION; i++) {
        if (hipMalloc(&oopStrategies[i],
                    params->oopSize * sizeof(char)) != hipSuccess) {
            printf("cuda malloc failed line 154\n");
            assert(0);
        }
    }
    if (hipMemcpy(cudaOopStrategies, oopStrategies,
            NUM_STRATEGIES_PER_ITERATION * sizeof(char*),
            hipMemcpyHostToDevice) != hipSuccess) {
        printf("cuda memcpy fialed line 160\n");
        assert(0);
    }

    int totalStrategies = 1;
    for (int i = 0; i < params->oopSize; i++) {
        totalStrategies *= OOP_MOVES;
    }
    output = (int *) malloc(NUM_STRATEGIES_PER_ITERATION * sizeof(int));
    if (hipMalloc(&cudaOutput,
                NUM_STRATEGIES_PER_ITERATION * sizeof(int)) != hipSuccess) {
        printf("cuda malloc failed line 172\n");
        assert(0);
    }

    char *curStrategy = (char *) malloc(params->oopSize * sizeof(char));
    memset(curStrategy, 0, params->oopSize * sizeof(char));

    int numThreads = MAX_THREADS > params->ipSize ? params->ipSize : MAX_THREADS;
    int numBlocks = MAX_BLOCKS > NUM_STRATEGIES_PER_ITERATION
        ? MAX_BLOCKS : NUM_STRATEGIES_PER_ITERATION;


    char *minStrategy = (char *) malloc(params->oopSize * sizeof(char));
    int minFound = INT_MAX;
    int numIter = totalStrategies / NUM_STRATEGIES_PER_ITERATION;
    if (numIter == 0) numIter = 1;
    clock_t startLoop = clock();
    clock_t start = clock();
    clock_t end;
    //number of kernel invokations needed
    for (int i = 0; i < numIter; i++) {
        //strategies per kernel call
        if (i>0 && i*NUM_STRATEGIES_PER_ITERATION%ITERATIONS_TO_PRINT==0) {
            end = clock();
            double time = (double) (end - start) / CLOCKS_PER_SEC;
            printf("Iteration: %d, Time: %.2f sec, Iterations per second: %.0f\n", i*NUM_STRATEGIES_PER_ITERATION, time, ITERATIONS_TO_PRINT / time);
            start = clock();
        }
        for (int j = 0; j < NUM_STRATEGIES_PER_ITERATION; j++) {
            addOne(curStrategy, params);
            if (hipMemcpy(oopStrategies[j], curStrategy, params->oopSize *
                        sizeof(char), hipMemcpyHostToDevice) != hipSuccess) {
                printf("CudaMemcpy Failed\n");
                assert(0);
            }
        }
        if (hipMemset(cudaOutput, 0,
                    NUM_STRATEGIES_PER_ITERATION * sizeof(int)) != hipSuccess) {
            printf("cuda memset failed line 197\n");
            assert(0);
        }
        kernel_calculateValue<<<numBlocks, numThreads>>>
            (numThreads, numBlocks, cudaOopStrategies, cudaOutput);
        if (hipMemcpy(output, cudaOutput, NUM_STRATEGIES_PER_ITERATION * sizeof(int),
                    hipMemcpyDeviceToHost) != hipSuccess) {
            printf("CudaMemcpy Failed\n");
            assert(0);
        }
        hipDeviceSynchronize();
        //need to synchronize here
        int minIdx = -1;
        //output is the value to the ip strategy, so find the minimum
        for (int k = 0; k < NUM_STRATEGIES_PER_ITERATION; k++) {
            if (output[k] < minFound) {
                minIdx = k;
                minFound = output[k];
            }
        }
        if (minIdx >= 0 && hipMemcpy(bestStrat, oopStrategies[minIdx],
                    params->oopSize * sizeof(char),
                    hipMemcpyDeviceToHost) != hipSuccess) {
            printf("CudaMemcpy Failed\n");
            assert(0);
        }
    }
    *stratVal = minFound;
    clock_t endLoop = clock();
    double time = (double) (endLoop - startLoop) / CLOCKS_PER_SEC;
    printf("Average iterations per second: %2.f\n", totalStrategies / time);
}
*/
extern "C"
void calcMaxIpStrategy(char *bestOopStrat, char *bestIpCheckStrat,
        char *bestIpBetStrat,GlobalConstants *params) {
    char *cudaOopStrat;
    char *cudaIpCheckStrat;
    char *cudaIpBetStrat;
    if (hipMalloc(&cudaOopStrat, params->oopSize * sizeof(char)) != hipSuccess) {
        printf("cuda malloc failed line 230\n");
        assert(0);
    }
    if (hipMemcpy(cudaOopStrat, bestOopStrat, params->oopSize * sizeof(char),
                hipMemcpyHostToDevice) != hipSuccess) {
        printf("cuda memcpy failed line 234");
        assert(0);
    }

    if (hipMalloc(&cudaIpCheckStrat, params->ipSize * sizeof(char)) != hipSuccess) {
        printf("cuda malloc failed line 239\n");
        assert(0);
    }
    if (hipMalloc(&cudaIpBetStrat, params->ipSize * sizeof(char)) != hipSuccess) {
        printf("cuda malloc failed line 239\n");
        assert(0);
    }

    int numThreads = MAX_THREADS > params->ipSize ? params->ipSize : MAX_THREADS;

    kernel_calculateIpStrat<<<1, numThreads>>>
        (numThreads, cudaOopStrat, cudaIpBetStrat, cudaIpCheckStrat);

    if (hipMemcpy(bestIpCheckStrat, cudaIpCheckStrat, params->ipSize * sizeof(char), hipMemcpyDeviceToHost) != hipSuccess) {
        printf("cuda memcpy failed line 299\n");
        assert(0);
    }
    if (hipMemcpy(bestIpBetStrat, cudaIpBetStrat, params->ipSize * sizeof(char), hipMemcpyDeviceToHost) != hipSuccess) {
        printf("cuda memcpy failed line 303\n");
        assert(0);
    }
}
