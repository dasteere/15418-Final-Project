#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <driver_functions.h>

struct GlobalConstants {
    __constant__ int *oopRange;
    __constant__ int oopSize;

    __constant__ int *ipRange;
    __constant__ int ipSize;

    __constant__ int potSize;
    __constant__ int betSize;
    __constant__ int afterBetSize;
};

#define CHECK_CALL 0
#define CHECK_FOLD 1
#define BET 2
#define OOP_MOVES 3

#define CHECK 4
#define BET 5
#define CALL 6
#define FOLD 7
#define IP_MOVES 4

#define NUM_STRATEGIES_PER_ITERATION 100

char **cudaOopStrategies;
int *cudaOutput;
int *output;

__global__ void kernel_calculateValue(int handsPerThread, int *output) {
    int idx = threadIdx.x;
    int strategyIdx = blockIdx.x;

    char *strategy = cudaOopStrategies[strategyIdx];
    int *out = output[strategyIdx];
    int check = 0;
    int bet = 0;
    int call = 0;
    int fold = 0;
    int cb_max = 0;
    int cf_max = 0;
    int ipRank, oopRank, oopMove, showdown, showPot, showBet;
    for (int i = idx; i < ipSize; i += handsPerThread) {
        int ipRank = ipRange[i];
        for (int j = 0; j < oopSize; j++) {
            int oopRank = oopRange[j];
            int oopMove = strategy[j];
            int showdown = ipRank > oopRank ? 1 : 0;
            showPot = showdown * potSize;
            showBet = showdown * afterBetSize;
            switch (oopMove) {
            case CHECK_CALL:
                check += showPot;
                bet += showBet;
            case CHECK_FOLD:
                check += showPot;
                bet += potSize;
            case BET:
                call += showdown * afterBetSize;
                fold -= potSize;
            }
        }
        cb_max = check > bet ? cc_check : cc_bet;
        cf_max = call > fold ? call : fold;
        //could be a bottleneck
        atomicAdd(output + strategyIdx, cb_max + cf_max);
        check = 0;
        bet = 0;
        call = 0;
        fold = 0;
    }
}

void addOne(int *curStrategy) {
    for (int i = 0; i < oopSize - 1; i++) {
        curStrategy[i] = (curStrategy[i] + 1) % OOP_MOVES;
        if (curStrategy[i] != 0) break;
    }
}

void calcMaxStrategy(char *bestStrat) {
    hipMalloc(&cudaOopStrategies, NUM_STRATEGIES_PER_ITERATION * sizeof(char *));
    for (int i = 0; i < NUM_STRATEGIES_PER_ITERATION; i++) {
        hipMalloc(&cudaOopStrategies[i], oopSize * sizeof(char));
    }
    int totalStrategies = 1;
    for (int i = 0; i < oopSize; i++) {
        totalStrategies *= OOP_MOVES;
    }
    output = (int *) malloc(NUM_STRATEGIES_PER_ITERATIONS * sizeof(int));
    char *curStrategy = (char *) malloc(oopSize * sizeof(char));
    memset(curStrategy, 0, oopSize * sizeof(char));
    hipMalloc(&cudaOutput, NUM_STRATEGIES_PER_ITERATION * sizeof(int));
    int numThreads = 64 > ipSize : ipSize ? 64;
    int handsPerThread = ipSize / numThreads;
    char *minStrategy = (char *) malloc(oopSize * sizeof(char));
    int minValue = 1 << (sizeof(int) - 2);
    for (int i = 0; i < totalStrategies / NUM_STRATEGIES_PER_ITERATION; i++) {
        for (int j = 0; j < NUM_STRATEGIES_PER_ITERATION; j++) {
            addOne(curStrategy);
            hipMemcpy(cudaOopStrategies[j], curStrategy, oopSize * sizeof(char), hipMemcpyHostToDevice);
        }
        kernel_calculateValue<<<NUM_STRATEGIES_PER_ITERATION, numThreads>>>(handsPerThread, cudaOutput);
        hipMemcpy(output, cudaOutput, NUM_THREADS_PER_ITERATION * sizeof(int), hipMemcpyDeviceToHost);
        //need to synchronize here
        int minIdx = -1;
        for (int k = 0; k < NUM_STRATEGIES_PER_ITERATION; k++) {
            if (output[k] < minFound) {
                minIdx = k;
                minFound = output[k];
            }
        }
        hipMemcpy(minStrategy, cudaOopStrategies[minIdx], oopSize * sizeof(char), hipMemcpyDeviceToHost);
    }
}
