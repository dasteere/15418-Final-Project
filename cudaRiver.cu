#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <stdio.h>
#include <limits.h>
#include <assert.h>


extern "C" {
#include "rank.h"
#include "cudaRiver.h"
#include "output_utils.h"
}

/*struct GlobalConstants {
    //must be in cuda
    int *oopRanks;
    int oopSize;

    //must be in cuda
    int *ipRanks;
    int ipSize;
    int potSize;
    int betSize;
    int afterBetSize;
};*/

__constant__ GlobalConstants cuConsts;

/*#define CHECK_CALL 0
#define CHECK_FOLD 1
#define BET 2
#define OOP_MOVES 3

#define IP_MOVES 4

#define NUM_STRATEGIES_PER_ITERATION 100*/

char **cudaOopStrategies;
int *cudaOutput;
int *output;

__global__ void kernel_calculateValue(int handsPerThread,
        char **cudaOopStrategies, int *output) {
    int idx = threadIdx.x;
    int strategyIdx = blockIdx.x;

    char *strategy = cudaOopStrategies[strategyIdx];
    int check = 0;
    int bet = 0;
    int call = 0;
    int fold = 0;
    int cb_max = 0;
    int cf_max = 0;
    int ipRank, oopRank, oopMove, showdown, showPot, showBet;
    for (int i = idx; i < cuConsts.ipSize; i += handsPerThread) {
        ipRank = cuConsts.ipRanks[i];
        for (int j = 0; j < cuConsts.oopSize; j++) {
            oopRank = cuConsts.oopRanks[j];
            oopMove = strategy[j];
            showdown = ipRank > oopRank ? 1 : -1;
            showPot = ipRank > oopRank ? cuConsts.potSize : 0;
            showBet = showPot + (showdown * cuConsts.betSize);
            switch (oopMove) {
            case CHECK_CALL:
                check += showPot;
                bet += showBet;
            case CHECK_FOLD:
                check += showPot;
                bet += cuConsts.potSize;
            case BET:
                call += showBet;
                fold -= cuConsts.potSize;
            }
        }
        cb_max = check > bet ? check : bet;
        cf_max = call > fold ? call : fold;
        //could be a bottleneck
        atomicAdd(output + strategyIdx, cb_max + cf_max);
        check = 0;
        bet = 0;
        call = 0;
        fold = 0;
    }
}

extern "C"
GlobalConstants *calcGlobalConsts(board_t board, hand_t *oopRange,
        int oopSize, hand_t *ipRange, int ipSize, int potSize, int betSize) {
    GlobalConstants *params = (GlobalConstants *) malloc(sizeof(GlobalConstants));
    int *oopRanks = (int *) malloc(oopSize * sizeof(int));
    int *ipRanks = (int *) malloc(ipSize * sizeof(int));
    for (int i = 0; i < oopSize; i++) {
        oopRanks[i] = rank_of(&board, &oopRange[i]);
    }
    for (int i = 0; i < ipSize; i++) {
        ipRanks[i] = rank_of(&board, &ipRange[i]);
    }
    if (hipMalloc(&(params->oopRanks), sizeof(int) * oopSize) != hipSuccess) {
        printf("Cuda malloc failed line 106\n");
        assert(0);
    }
    hipMalloc(&(params->ipRanks), sizeof(int) * ipSize);

    if (hipMemcpy(params->oopRanks, oopRanks, sizeof(int) * oopSize,
                hipMemcpyHostToDevice) != hipSuccess) {
        printf("Cuda Memcpy failed line 109\n");
        assert(0);
    }
    if (hipMemcpy(params->ipRanks, ipRanks, sizeof(int) * ipSize,
                hipMemcpyHostToDevice) != hipSuccess) {
        printf("Cuda Memcpy Failed line 113\n");
        assert(0);
    }

    params->oopSize = oopSize;
    params->ipSize = ipSize;
    params->potSize = potSize;
    params->betSize = betSize;
    params->afterBetSize = potSize + betSize;

    return params;
}

void addOne(char *curStrategy, GlobalConstants *params) {
    for (int i = 0; i < params->oopSize - 1; i++) {
        curStrategy[i] = (curStrategy[i] + 1) % OOP_MOVES;
        if (curStrategy[i] != 0) break;
    }
}

//calculates the best strategy for the oop player along with the strategies value
extern "C"
void calcMaxStrategy(char *bestStrat, int *stratVal, GlobalConstants *params) {
    if (hipMemcpyToSymbol(HIP_SYMBOL(cuConsts), params,
                sizeof(GlobalConstants)) != hipSuccess) {
        printf("cuda memcpy to symbol failed line 141\n");
        assert(0);
    }


    char **oopStrategies =
        (char **) malloc(NUM_STRATEGIES_PER_ITERATION * sizeof(char *));
    if (hipMalloc(&cudaOopStrategies,
                NUM_STRATEGIES_PER_ITERATION * sizeof(char *)) != hipSuccess) {
        printf("Cuda malloc failed line 149\n");
        assert(0);
    }

    for (int i = 0; i < NUM_STRATEGIES_PER_ITERATION; i++) {
        if (hipMalloc(&oopStrategies[i],
                    params->oopSize * sizeof(char)) != hipSuccess) {
            printf("cuda malloc failed line 154\n");
            assert(0);
        }
    }
    if (hipMemcpy(cudaOopStrategies, oopStrategies,
            NUM_STRATEGIES_PER_ITERATION * sizeof(char*),
            hipMemcpyHostToDevice) != hipSuccess) {
        printf("cuda memcpy fialed line 160\n");
        assert(0);
    }

    int totalStrategies = 1;
    for (int i = 0; i < params->oopSize; i++) {
        totalStrategies *= OOP_MOVES;
    }

    output = (int *) malloc(NUM_STRATEGIES_PER_ITERATION * sizeof(int));
    if (hipMalloc(&cudaOutput,
                NUM_STRATEGIES_PER_ITERATION * sizeof(int)) != hipSuccess) {
        printf("cuda malloc failed line 172\n");
        assert(0);
    }

    char *curStrategy = (char *) malloc(params->oopSize * sizeof(char));
    memset(curStrategy, 0, params->oopSize * sizeof(char));

    int numThreads = 64 > params->ipSize ? params->ipSize : 64;

    int handsPerThread = params->ipSize / numThreads;
    char *minStrategy = (char *) malloc(params->oopSize * sizeof(char));
    int minFound = INT_MAX;

    //number of kernel invokations needed
    for (int i = 0; i < totalStrategies / NUM_STRATEGIES_PER_ITERATION; i++) {
        //strategies per kernel call
        for (int j = 0; j < NUM_STRATEGIES_PER_ITERATION; j++) {
            addOne(curStrategy, params);
            if (hipMemcpy(oopStrategies[j], curStrategy, params->oopSize *
                        sizeof(char), hipMemcpyHostToDevice) != hipSuccess) {
                printf("CudaMemcpy Failed\n");
                assert(0);
            }
        }
        if (hipMemset(cudaOutput, 0,
                    NUM_STRATEGIES_PER_ITERATION * sizeof(int)) != hipSuccess) {
            printf("cuda memset failed line 197\n");
            assert(0);
        }
        kernel_calculateValue<<<NUM_STRATEGIES_PER_ITERATION, numThreads>>>
            (handsPerThread, cudaOopStrategies, cudaOutput);
        if (hipMemcpy(output, cudaOutput, NUM_STRATEGIES_PER_ITERATION * sizeof(int),
                    hipMemcpyDeviceToHost) != hipSuccess) {
            printf("CudaMemcpy Failed\n");
            assert(0);
        }

        //need to synchronize here
        int minIdx = -1;
        //output is the value to the ip strategy, so find the minimum
        for (int k = 0; k < NUM_STRATEGIES_PER_ITERATION; k++) {
            if (output[k] < minFound) {
                minIdx = k;
                minFound = output[k];
            }
        }
        if (minIdx >= 0 && hipMemcpy(bestStrat, oopStrategies[minIdx],
                    params->oopSize * sizeof(char),
                    hipMemcpyDeviceToHost) != hipSuccess) {
            printf("CudaMemcpy Failed\n");
            assert(0);
        }
    }
    *stratVal = minFound;
}
